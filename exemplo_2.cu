
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel CUDA para somar dois vetores
__global__ void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1<<20; // 1 milhão de elementos
    float *x, *y, *d_x, *d_y;

    // Aloca memória no host
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    // Inicializa os vetores no host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Aloca memória na GPU
    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    // Copia os vetores para a GPU
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Executa o kernel add() na GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, d_x, d_y);

    // Copia os resultados de volta para o host
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    printf("y[0] = %f\n", y[0]);

    // Libera a memória
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    
    return 0;
}
